#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 32

extern "C" void deconvolve (int N1, int N2, double *uIni, double *srcImg, int itertime, double h, double lambda, double delta, double epsilon, double * dataNow);

__global__ void nablaIni_kernel(int N1, int N2, double *nablaU){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  if (i<N1&&j<N2){
    nablaU[i*N2+j]=0;
  }
}

__global__ void normNabla_kernel(int N1, int N2, double *u, double h, double * nablaU){
  double ex, ey;
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<N1-1&&j<N2-1&&i>0&&j>0){
    ex=(u[(i+1)*N2+j]-u[(i-1)*N2+j])/2/h;
    ey=(u[i*N2+j+1]-u[i*N2+j-1])/2/h;
    nablaU[i*N2+j]=sqrt(ex*ex+ey*ey);
  }
}

__global__ void normMinus_kernel(int N1, int N2, double * a, double * b, double * result){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<N1&&j<N2){
    result[i*N2+j]=abs(a[i*N2+j]-b[i*N2+j]);
  }
}

__global__ void fcal_kernel(int N1, int N2, double *srcImg, double *f, double *nablaU, double *uMinusf, double lambda, double delta, double epsilon){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<N1&&j<N2){
    f[i*N2+j]=lambda*srcImg[i*N2+j]/sqrt(uMinusf[i*N2+j]*uMinusf[i*N2+j]+delta)*sqrt(nablaU[i*N2+j]*nablaU[i*N2+j]+epsilon);
  }
}

__global__ void deblur_kernel(int N1, int N2, double *u, double *srcImg, double *deblurU, double *f, double *nablaU, double *uMinusf, double h, double lambda, double delta, double epsilon){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;

  if (i<N1-1&&j<N2-1&&i>0&&j>0){
    deblurU[i*N2+j]=(h*h*f[i*N2+j]+u[(i-1)*N2+j]+u[i*N2+j-1]+u[(i+1)*N2+j]+u[i*N2+j+1])/(4+lambda*sqrt(nablaU[i*N2+j]*nablaU[i*N2+j]+epsilon)*h*h/sqrt(uMinusf[i*N2+j]*uMinusf[i*N2+j]+delta));
    if (deblurU[i*N2+j]>255){deblurU[i*N2+j]=255;}
    if (deblurU[i*N2+j]<0){deblurU[i*N2+j]=0;}
  }
}

__global__ void deblur_kernel(int N1, int N2, double *nabla){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  if(i<N1&&j<N2){
    nabla[i*N2+j]=0;
  }
}

double residual(int N1, int N2, double* u, double* f) {
        double residual_norm_sq = 0.0;
        for (int i = 0; i < N1 * N2; i++) {
        residual_norm_sq += (u[i] - f[i]) * (u[i] - f[i]);
        }
        return sqrt(residual_norm_sq);
}

void deconvolve (int N1, int N2, double *uIni, double *srcImg, int itertime, double h, double lambda, double delta, double epsilon, double * u) {
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(N1/BLOCK_SIZE+1,N2/BLOCK_SIZE+1);

  double *u_d,*u_prev_d, *f, *nablaU, *uMinusf;
  double *srcImg_d;
  hipMalloc(&u_d,N1*N2*sizeof(double));
  hipMalloc(&u_prev_d,N1*N2*sizeof(double));
  hipMalloc(&f,N1*N2*sizeof(double));
  hipMalloc(&nablaU,N1*N2*sizeof(double));
  hipMalloc(&uMinusf,N1*N2*sizeof(double));
  hipMalloc(&srcImg_d, N1*N2*sizeof(double));

  hipMemcpy(srcImg_d, srcImg, N1*N2*sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(u_d,uIni,N1*N2*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(u_prev_d,uIni,N1*N2*sizeof(double),hipMemcpyHostToDevice);

  clock_t t;
  t=clock();
  printf("Initial residule: %f\n", residual(N1,N2, uIni, srcImg));
  printf("time= %f seconds\n", ((float)t)/CLOCKS_PER_SEC);
  for (int i=0; i<itertime;i++){
    //printf("itertime=%d\n",i);
    nablaIni_kernel<<<dimGrid,dimBlock>>>(N1,N2,nablaU);
    normNabla_kernel<<<dimGrid,dimBlock>>>(N1, N2, u_d, h, nablaU);
    normMinus_kernel<<<dimGrid,dimBlock>>>(N1, N2, u_d, srcImg_d, uMinusf);
    fcal_kernel<<<dimGrid,dimBlock>>>(N1,N2,srcImg_d,f,nablaU,uMinusf,lambda, delta,epsilon);
    deblur_kernel<<<dimGrid,dimBlock>>>(N1,N2, u_prev_d,srcImg_d,u_d,f, nablaU, uMinusf, h, lambda, delta, epsilon);
    hipMemcpy(u_prev_d,u_d,N1*N2*sizeof(double),hipMemcpyDeviceToDevice);
  }
  hipMemcpy(u,u_prev_d,N1*N2*sizeof(double),hipMemcpyDeviceToHost);

  //cudaMemcpy(u, srcImg_d, N1*N2*sizeof(double), cudaMemcpyDeviceToHost);
  
  t=clock()-t;
  printf("Final residule: %f\n", residual(N1,N2, u, srcImg));
  printf("time= %f seconds\n", ((float)t)/CLOCKS_PER_SEC);
  hipFree(u_d);
  hipFree(u_prev_d);
  hipFree(f);
  hipFree(nablaU);
  hipFree(uMinusf);
  hipFree(srcImg_d);
}

